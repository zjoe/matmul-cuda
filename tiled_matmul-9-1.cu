#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>

#define TILE_WIDTH 64
#define WIDTH_PER_THREAD 8
#define SW TILE_WIDTH/WIDTH_PER_THREAD
#define N 2048

texture<float, 2, hipReadModeElementType> tex_A;
texture<float, 2, hipReadModeElementType> tex_B;
surface<void, 2> surf_C;

void err_handling(hipError_t *err, const char *str)
{
	if (*err != hipSuccess) {
		printf("%s\n", str);
		exit(EXIT_FAILURE);
	}
}

__global__ void matMul(const float *A, const float *B, float *C, int m, int k, int n)
{
	__shared__ float sA_bf[2][8*64];
	__shared__ float sB_bf[2][8*64];
	float *A_pref, *A_now;
	float *B_pref, *B_now;

	int x = threadIdx.x;
	int y = threadIdx.y;

	int bx = blockIdx.x*64;
	int by = blockIdx.y*64;
	
	int id = y*8+x;
	int inv_id = (id%32)/4*8 + id%4 + (id < 32 ? 0 : 4);
	int glbA_id = by + inv_id;
	int glbB_id = bx + inv_id;

	int row = by + y*8;
	int col = bx + x*8;


	float a0[8];
	float a1[8];
	float b0[8];
	float b1[8];

	float c00 = 0.0; float c01 = 0.0; float c02 = 0.0; float c03 = 0.0, c04 = 0.0; float c05 = 0.0; float c06 = 0.0; float c07 = 0.0;
	float c10 = 0.0; float c11 = 0.0; float c12 = 0.0; float c13 = 0.0, c14 = 0.0; float c15 = 0.0; float c16 = 0.0; float c17 = 0.0;
	float c20 = 0.0; float c21 = 0.0; float c22 = 0.0; float c23 = 0.0, c24 = 0.0; float c25 = 0.0; float c26 = 0.0; float c27 = 0.0;
	float c30 = 0.0; float c31 = 0.0; float c32 = 0.0; float c33 = 0.0, c34 = 0.0; float c35 = 0.0; float c36 = 0.0; float c37 = 0.0;
	float c40 = 0.0; float c41 = 0.0; float c42 = 0.0; float c43 = 0.0, c44 = 0.0; float c45 = 0.0; float c46 = 0.0; float c47 = 0.0;
	float c50 = 0.0; float c51 = 0.0; float c52 = 0.0; float c53 = 0.0, c54 = 0.0; float c55 = 0.0; float c56 = 0.0; float c57 = 0.0;
	float c60 = 0.0; float c61 = 0.0; float c62 = 0.0; float c63 = 0.0, c64 = 0.0; float c65 = 0.0; float c66 = 0.0; float c67 = 0.0;
	float c70 = 0.0; float c71 = 0.0; float c72 = 0.0; float c73 = 0.0, c74 = 0.0; float c75 = 0.0; float c76 = 0.0; float c77 = 0.0;

	sA_bf[0][0*64+id] = tex2D(tex_A, 0, glbA_id);
	sA_bf[0][1*64+id] = tex2D(tex_A, 1, glbA_id);
	sA_bf[0][2*64+id] = tex2D(tex_A, 2, glbA_id);
	sA_bf[0][3*64+id] = tex2D(tex_A, 3, glbA_id);
	sA_bf[0][4*64+id] = tex2D(tex_A, 4, glbA_id);
	sA_bf[0][5*64+id] = tex2D(tex_A, 5, glbA_id);
	sA_bf[0][6*64+id] = tex2D(tex_A, 6, glbA_id);
	sA_bf[0][7*64+id] = tex2D(tex_A, 7, glbA_id);

	sB_bf[0][0*64+id] = tex2D(tex_B, glbB_id, 0);
	sB_bf[0][1*64+id] = tex2D(tex_B, glbB_id, 1);
	sB_bf[0][2*64+id] = tex2D(tex_B, glbB_id, 2);
	sB_bf[0][3*64+id] = tex2D(tex_B, glbB_id, 3);
	sB_bf[0][4*64+id] = tex2D(tex_B, glbB_id, 4);
	sB_bf[0][5*64+id] = tex2D(tex_B, glbB_id, 5);
	sB_bf[0][6*64+id] = tex2D(tex_B, glbB_id, 6);
	sB_bf[0][7*64+id] = tex2D(tex_B, glbB_id, 7);

	A_pref = sA_bf[1];
	B_pref = sB_bf[1];
	A_now  = sA_bf[0];
	B_now  = sB_bf[0];

	int track_bf = 0;

	for (int t = 8; t < k; t += 8) {

		__syncthreads();

		A_pref[id] = tex2D(tex_A, t, glbA_id);
		B_pref[id] = tex2D(tex_B, glbB_id, t);


		((float4*)a0)[0] = ((float4*)A_now)[y];
		((float4*)a0)[1] = ((float4*)A_now)[y+8];
		
		((float4*)b0)[0] = ((float4*)B_now)[x];
		((float4*)b0)[1] = ((float4*)B_now)[x+8];
		

		#pragma unroll
		for (int i = 1; i < 8; ++i) {
			int base = i * 16;
			A_pref[i*64+id] = tex2D(tex_A, t+i, glbA_id);
			B_pref[i*64+id] = tex2D(tex_B, glbB_id, t+i);

			if (i&1) {
				c00 += a0[0] * b0[0];
				c01 += a0[0] * b0[1];
				c02 += a0[0] * b0[2];
				c03 += a0[0] * b0[3];
				c04 += a0[0] * b0[4];
				c05 += a0[0] * b0[5];
				c06 += a0[0] * b0[6];
				c07 += a0[0] * b0[7];
				((float4*)a1)[0] = ((float4*)A_now)[base+y];

				c10 += a0[1] * b0[0];
				c11 += a0[1] * b0[1];
				c12 += a0[1] * b0[2];
				c13 += a0[1] * b0[3];
				c14 += a0[1] * b0[4];
				c15 += a0[1] * b0[5];
				c16 += a0[1] * b0[6];
				c17 += a0[1] * b0[7];

				c20 += a0[2] * b0[0];
				c21 += a0[2] * b0[1];
				c22 += a0[2] * b0[2];
				c23 += a0[2] * b0[3];
				c24 += a0[2] * b0[4];
				c25 += a0[2] * b0[5];
				c26 += a0[2] * b0[6];
				c27 += a0[2] * b0[7];
				((float4*)b1)[0] = ((float4*)B_now)[base+x];

				c30 += a0[3] * b0[0];
				c31 += a0[3] * b0[1];
				c32 += a0[3] * b0[2];
				c33 += a0[3] * b0[3];
				c34 += a0[3] * b0[4];
				c35 += a0[3] * b0[5];
				c36 += a0[3] * b0[6];
				c37 += a0[3] * b0[7];

				c40 += a0[4] * b0[0];
				c41 += a0[4] * b0[1];
				c42 += a0[4] * b0[2];
				c43 += a0[4] * b0[3];
				c44 += a0[4] * b0[4];
				c45 += a0[4] * b0[5];
				c46 += a0[4] * b0[6];
				c47 += a0[4] * b0[7];
				((float4*)a1)[1] = ((float4*)A_now)[base+y+8];

				c50 += a0[5] * b0[0];
				c51 += a0[5] * b0[1];
				c52 += a0[5] * b0[2];
				c53 += a0[5] * b0[3];
				c54 += a0[5] * b0[4];
				c55 += a0[5] * b0[5];
				c56 += a0[5] * b0[6];
				c57 += a0[5] * b0[7];

				c60 += a0[6] * b0[0];
				c61 += a0[6] * b0[1];
				c62 += a0[6] * b0[2];
				c63 += a0[6] * b0[3];
				c64 += a0[6] * b0[4];
				c65 += a0[6] * b0[5];
				c66 += a0[6] * b0[6];
				c67 += a0[6] * b0[7];
				((float4*)b1)[1] = ((float4*)B_now)[base+x+8];

				c70 += a0[7] * b0[0];
				c71 += a0[7] * b0[1];
				c72 += a0[7] * b0[2];
				c73 += a0[7] * b0[3];
				c74 += a0[7] * b0[4];
				c75 += a0[7] * b0[5];
				c76 += a0[7] * b0[6];
				c77 += a0[7] * b0[7];
			} else {
				c00 += a1[0] * b1[0];
				c01 += a1[0] * b1[1];
				c02 += a1[0] * b1[2];
				c03 += a1[0] * b1[3];
				c04 += a1[0] * b1[4];
				c05 += a1[0] * b1[5];
				c06 += a1[0] * b1[6];
				c07 += a1[0] * b1[7];
				((float4*)a0)[0] = ((float4*)A_now)[base+y];

				c10 += a1[1] * b1[0];
				c11 += a1[1] * b1[1];
				c12 += a1[1] * b1[2];
				c13 += a1[1] * b1[3];
				c14 += a1[1] * b1[4];
				c15 += a1[1] * b1[5];
				c16 += a1[1] * b1[6];
				c17 += a1[1] * b1[7];

				c20 += a1[2] * b1[0];
				c21 += a1[2] * b1[1];
				c22 += a1[2] * b1[2];
				c23 += a1[2] * b1[3];
				c24 += a1[2] * b1[4];
				c25 += a1[2] * b1[5];
				c26 += a1[2] * b1[6];
				c27 += a1[2] * b1[7];
				((float4*)b0)[0] = ((float4*)B_now)[base+x];

				c30 += a1[3] * b1[0];
				c31 += a1[3] * b1[1];
				c32 += a1[3] * b1[2];
				c33 += a1[3] * b1[3];
				c34 += a1[3] * b1[4];
				c35 += a1[3] * b1[5];
				c36 += a1[3] * b1[6];
				c37 += a1[3] * b1[7];

				c40 += a1[4] * b1[0];
				c41 += a1[4] * b1[1];
				c42 += a1[4] * b1[2];
				c43 += a1[4] * b1[3];
				c44 += a1[4] * b1[4];
				c45 += a1[4] * b1[5];
				c46 += a1[4] * b1[6];
				c47 += a1[4] * b1[7];
				((float4*)a0)[1] = ((float4*)A_now)[base+y+8];

				c50 += a1[5] * b1[0];
				c51 += a1[5] * b1[1];
				c52 += a1[5] * b1[2];
				c53 += a1[5] * b1[3];
				c54 += a1[5] * b1[4];
				c55 += a1[5] * b1[5];
				c56 += a1[5] * b1[6];
				c57 += a1[5] * b1[7];

				c60 += a1[6] * b1[0];
				c61 += a1[6] * b1[1];
				c62 += a1[6] * b1[2];
				c63 += a1[6] * b1[3];
				c64 += a1[6] * b1[4];
				c65 += a1[6] * b1[5];
				c66 += a1[6] * b1[6];
				c67 += a1[6] * b1[7];
				((float4*)b0)[1] = ((float4*)B_now)[base+x+8];

				c70 += a1[7] * b1[0];
				c71 += a1[7] * b1[1];
				c72 += a1[7] * b1[2];
				c73 += a1[7] * b1[3];
				c74 += a1[7] * b1[4];
				c75 += a1[7] * b1[5];
				c76 += a1[7] * b1[6];
				c77 += a1[7] * b1[7];
			}
		}
		c00 += a1[0] * b1[0];
		c01 += a1[0] * b1[1];
		c02 += a1[0] * b1[2];
		c03 += a1[0] * b1[3];
		c04 += a1[0] * b1[4];
		c05 += a1[0] * b1[5];
		c06 += a1[0] * b1[6];
		c07 += a1[0] * b1[7];

		c10 += a1[1] * b1[0];
		c11 += a1[1] * b1[1];
		c12 += a1[1] * b1[2];
		c13 += a1[1] * b1[3];
		c14 += a1[1] * b1[4];
		c15 += a1[1] * b1[5];
		c16 += a1[1] * b1[6];
		c17 += a1[1] * b1[7];

		c20 += a1[2] * b1[0];
		c21 += a1[2] * b1[1];
		c22 += a1[2] * b1[2];
		c23 += a1[2] * b1[3];
		c24 += a1[2] * b1[4];
		c25 += a1[2] * b1[5];
		c26 += a1[2] * b1[6];
		c27 += a1[2] * b1[7];

		c30 += a1[3] * b1[0];
		c31 += a1[3] * b1[1];
		c32 += a1[3] * b1[2];
		c33 += a1[3] * b1[3];
		c34 += a1[3] * b1[4];
		c35 += a1[3] * b1[5];
		c36 += a1[3] * b1[6];
		c37 += a1[3] * b1[7];

		c40 += a1[4] * b1[0];
		c41 += a1[4] * b1[1];
		c42 += a1[4] * b1[2];
		c43 += a1[4] * b1[3];
		c44 += a1[4] * b1[4];
		c45 += a1[4] * b1[5];
		c46 += a1[4] * b1[6];
		c47 += a1[4] * b1[7];

		c50 += a1[5] * b1[0];
		c51 += a1[5] * b1[1];
		c52 += a1[5] * b1[2];
		c53 += a1[5] * b1[3];
		c54 += a1[5] * b1[4];
		c55 += a1[5] * b1[5];
		c56 += a1[5] * b1[6];
		c57 += a1[5] * b1[7];

		c60 += a1[6] * b1[0];
		c61 += a1[6] * b1[1];
		c62 += a1[6] * b1[2];
		c63 += a1[6] * b1[3];
		c64 += a1[6] * b1[4];
		c65 += a1[6] * b1[5];
		c66 += a1[6] * b1[6];
		c67 += a1[6] * b1[7];

		c70 += a1[7] * b1[0];
		c71 += a1[7] * b1[1];
		c72 += a1[7] * b1[2];
		c73 += a1[7] * b1[3];
		c74 += a1[7] * b1[4];
		c75 += a1[7] * b1[5];
		c76 += a1[7] * b1[6];
		c77 += a1[7] * b1[7];

		A_pref = sA_bf[track_bf];
		B_pref = sB_bf[track_bf];
		A_now  = sA_bf[1-track_bf];
		B_now  = sB_bf[1-track_bf];
		track_bf = 1 - track_bf;

	}
	__syncthreads();


	((float4*)a0)[0] = ((float4*)A_now)[y];
	((float4*)a0)[1] = ((float4*)A_now)[y+8];
	
	((float4*)b0)[0] = ((float4*)B_now)[x];
	((float4*)b0)[1] = ((float4*)B_now)[x+8];

	#pragma unroll
	for (int i = 1; i < 8; ++i) {
		int base = i * 16;

		if (i&1) {
			c00 += a0[0] * b0[0];
			c01 += a0[0] * b0[1];
			c02 += a0[0] * b0[2];
			c03 += a0[0] * b0[3];
			c04 += a0[0] * b0[4];
			c05 += a0[0] * b0[5];
			c06 += a0[0] * b0[6];
			c07 += a0[0] * b0[7];
			((float4*)a1)[0] = ((float4*)A_now)[base+y];

			c10 += a0[1] * b0[0];
			c11 += a0[1] * b0[1];
			c12 += a0[1] * b0[2];
			c13 += a0[1] * b0[3];
			c14 += a0[1] * b0[4];
			c15 += a0[1] * b0[5];
			c16 += a0[1] * b0[6];
			c17 += a0[1] * b0[7];

			c20 += a0[2] * b0[0];
			c21 += a0[2] * b0[1];
			c22 += a0[2] * b0[2];
			c23 += a0[2] * b0[3];
			c24 += a0[2] * b0[4];
			c25 += a0[2] * b0[5];
			c26 += a0[2] * b0[6];
			c27 += a0[2] * b0[7];
			((float4*)b1)[0] = ((float4*)B_now)[base+x];

			c30 += a0[3] * b0[0];
			c31 += a0[3] * b0[1];
			c32 += a0[3] * b0[2];
			c33 += a0[3] * b0[3];
			c34 += a0[3] * b0[4];
			c35 += a0[3] * b0[5];
			c36 += a0[3] * b0[6];
			c37 += a0[3] * b0[7];

			c40 += a0[4] * b0[0];
			c41 += a0[4] * b0[1];
			c42 += a0[4] * b0[2];
			c43 += a0[4] * b0[3];
			c44 += a0[4] * b0[4];
			c45 += a0[4] * b0[5];
			c46 += a0[4] * b0[6];
			c47 += a0[4] * b0[7];
			((float4*)a1)[1] = ((float4*)A_now)[base+y+8];

			c50 += a0[5] * b0[0];
			c51 += a0[5] * b0[1];
			c52 += a0[5] * b0[2];
			c53 += a0[5] * b0[3];
			c54 += a0[5] * b0[4];
			c55 += a0[5] * b0[5];
			c56 += a0[5] * b0[6];
			c57 += a0[5] * b0[7];

			c60 += a0[6] * b0[0];
			c61 += a0[6] * b0[1];
			c62 += a0[6] * b0[2];
			c63 += a0[6] * b0[3];
			c64 += a0[6] * b0[4];
			c65 += a0[6] * b0[5];
			c66 += a0[6] * b0[6];
			c67 += a0[6] * b0[7];
			((float4*)b1)[1] = ((float4*)B_now)[base+x+8];

			c70 += a0[7] * b0[0];
			c71 += a0[7] * b0[1];
			c72 += a0[7] * b0[2];
			c73 += a0[7] * b0[3];
			c74 += a0[7] * b0[4];
			c75 += a0[7] * b0[5];
			c76 += a0[7] * b0[6];
			c77 += a0[7] * b0[7];
		} else {
			c00 += a1[0] * b1[0];
			c01 += a1[0] * b1[1];
			c02 += a1[0] * b1[2];
			c03 += a1[0] * b1[3];
			c04 += a1[0] * b1[4];
			c05 += a1[0] * b1[5];
			c06 += a1[0] * b1[6];
			c07 += a1[0] * b1[7];
			((float4*)a0)[0] = ((float4*)A_now)[base+y];

			c10 += a1[1] * b1[0];
			c11 += a1[1] * b1[1];
			c12 += a1[1] * b1[2];
			c13 += a1[1] * b1[3];
			c14 += a1[1] * b1[4];
			c15 += a1[1] * b1[5];
			c16 += a1[1] * b1[6];
			c17 += a1[1] * b1[7];

			c20 += a1[2] * b1[0];
			c21 += a1[2] * b1[1];
			c22 += a1[2] * b1[2];
			c23 += a1[2] * b1[3];
			c24 += a1[2] * b1[4];
			c25 += a1[2] * b1[5];
			c26 += a1[2] * b1[6];
			c27 += a1[2] * b1[7];
			((float4*)b0)[0] = ((float4*)B_now)[base+x];

			c30 += a1[3] * b1[0];
			c31 += a1[3] * b1[1];
			c32 += a1[3] * b1[2];
			c33 += a1[3] * b1[3];
			c34 += a1[3] * b1[4];
			c35 += a1[3] * b1[5];
			c36 += a1[3] * b1[6];
			c37 += a1[3] * b1[7];

			c40 += a1[4] * b1[0];
			c41 += a1[4] * b1[1];
			c42 += a1[4] * b1[2];
			c43 += a1[4] * b1[3];
			c44 += a1[4] * b1[4];
			c45 += a1[4] * b1[5];
			c46 += a1[4] * b1[6];
			c47 += a1[4] * b1[7];
			((float4*)a0)[1] = ((float4*)A_now)[base+y+8];

			c50 += a1[5] * b1[0];
			c51 += a1[5] * b1[1];
			c52 += a1[5] * b1[2];
			c53 += a1[5] * b1[3];
			c54 += a1[5] * b1[4];
			c55 += a1[5] * b1[5];
			c56 += a1[5] * b1[6];
			c57 += a1[5] * b1[7];

			c60 += a1[6] * b1[0];
			c61 += a1[6] * b1[1];
			c62 += a1[6] * b1[2];
			c63 += a1[6] * b1[3];
			c64 += a1[6] * b1[4];
			c65 += a1[6] * b1[5];
			c66 += a1[6] * b1[6];
			c67 += a1[6] * b1[7];
			((float4*)b0)[1] = ((float4*)B_now)[base+x+8];

			c70 += a1[7] * b1[0];
			c71 += a1[7] * b1[1];
			c72 += a1[7] * b1[2];
			c73 += a1[7] * b1[3];
			c74 += a1[7] * b1[4];
			c75 += a1[7] * b1[5];
			c76 += a1[7] * b1[6];
			c77 += a1[7] * b1[7];
		}
	}
	c00 += a1[0] * b1[0];
	c01 += a1[0] * b1[1];
	c02 += a1[0] * b1[2];
	c03 += a1[0] * b1[3];
	c04 += a1[0] * b1[4];
	c05 += a1[0] * b1[5];
	c06 += a1[0] * b1[6];
	c07 += a1[0] * b1[7];

	c10 += a1[1] * b1[0];
	c11 += a1[1] * b1[1];
	c12 += a1[1] * b1[2];
	c13 += a1[1] * b1[3];
	c14 += a1[1] * b1[4];
	c15 += a1[1] * b1[5];
	c16 += a1[1] * b1[6];
	c17 += a1[1] * b1[7];

	c20 += a1[2] * b1[0];
	c21 += a1[2] * b1[1];
	c22 += a1[2] * b1[2];
	c23 += a1[2] * b1[3];
	c24 += a1[2] * b1[4];
	c25 += a1[2] * b1[5];
	c26 += a1[2] * b1[6];
	c27 += a1[2] * b1[7];

	c30 += a1[3] * b1[0];
	c31 += a1[3] * b1[1];
	c32 += a1[3] * b1[2];
	c33 += a1[3] * b1[3];
	c34 += a1[3] * b1[4];
	c35 += a1[3] * b1[5];
	c36 += a1[3] * b1[6];
	c37 += a1[3] * b1[7];

	c40 += a1[4] * b1[0];
	c41 += a1[4] * b1[1];
	c42 += a1[4] * b1[2];
	c43 += a1[4] * b1[3];
	c44 += a1[4] * b1[4];
	c45 += a1[4] * b1[5];
	c46 += a1[4] * b1[6];
	c47 += a1[4] * b1[7];

	c50 += a1[5] * b1[0];
	c51 += a1[5] * b1[1];
	c52 += a1[5] * b1[2];
	c53 += a1[5] * b1[3];
	c54 += a1[5] * b1[4];
	c55 += a1[5] * b1[5];
	c56 += a1[5] * b1[6];
	c57 += a1[5] * b1[7];

	c60 += a1[6] * b1[0];
	c61 += a1[6] * b1[1];
	c62 += a1[6] * b1[2];
	c63 += a1[6] * b1[3];
	c64 += a1[6] * b1[4];
	c65 += a1[6] * b1[5];
	c66 += a1[6] * b1[6];
	c67 += a1[6] * b1[7];

	c70 += a1[7] * b1[0];
	c71 += a1[7] * b1[1];
	c72 += a1[7] * b1[2];
	c73 += a1[7] * b1[3];
	c74 += a1[7] * b1[4];
	c75 += a1[7] * b1[5];
	c76 += a1[7] * b1[6];
	c77 += a1[7] * b1[7];


/*
	for (int i = 0; i < 7; i++) {
		for (int j = 0; j < 7; j++) {
			surf2Dwrite(c00, surf_C, (col  )*sizeof(float), row  , hipBoundaryModeZero);
		}
	}
*/
	surf2Dwrite(c00, surf_C, (col  )*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c01, surf_C, (col+1)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c02, surf_C, (col+2)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c03, surf_C, (col+3)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c04, surf_C, (col+4)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c05, surf_C, (col+5)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c06, surf_C, (col+6)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c07, surf_C, (col+7)*sizeof(float), row  , hipBoundaryModeZero);

	surf2Dwrite(c10, surf_C, (col  )*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c11, surf_C, (col+1)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c12, surf_C, (col+2)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c13, surf_C, (col+3)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c14, surf_C, (col+4)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c15, surf_C, (col+5)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c16, surf_C, (col+6)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c17, surf_C, (col+7)*sizeof(float), row+1  , hipBoundaryModeZero);

	surf2Dwrite(c20, surf_C, (col  )*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c21, surf_C, (col+1)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c22, surf_C, (col+2)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c23, surf_C, (col+3)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c24, surf_C, (col+4)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c25, surf_C, (col+5)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c26, surf_C, (col+6)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c27, surf_C, (col+7)*sizeof(float), row+2  , hipBoundaryModeZero);

	surf2Dwrite(c30, surf_C, (col  )*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c31, surf_C, (col+1)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c32, surf_C, (col+2)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c33, surf_C, (col+3)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c34, surf_C, (col+4)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c35, surf_C, (col+5)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c36, surf_C, (col+6)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c37, surf_C, (col+7)*sizeof(float), row+3  , hipBoundaryModeZero);

	surf2Dwrite(c40, surf_C, (col  )*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c41, surf_C, (col+1)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c42, surf_C, (col+2)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c43, surf_C, (col+3)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c44, surf_C, (col+4)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c45, surf_C, (col+5)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c46, surf_C, (col+6)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c47, surf_C, (col+7)*sizeof(float), row+4  , hipBoundaryModeZero);

	surf2Dwrite(c50, surf_C, (col  )*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c51, surf_C, (col+1)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c52, surf_C, (col+2)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c53, surf_C, (col+3)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c54, surf_C, (col+4)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c55, surf_C, (col+5)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c56, surf_C, (col+6)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c57, surf_C, (col+7)*sizeof(float), row+5  , hipBoundaryModeZero);

	surf2Dwrite(c60, surf_C, (col  )*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c61, surf_C, (col+1)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c62, surf_C, (col+2)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c63, surf_C, (col+3)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c64, surf_C, (col+4)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c65, surf_C, (col+5)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c66, surf_C, (col+6)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c67, surf_C, (col+7)*sizeof(float), row+6  , hipBoundaryModeZero);

	surf2Dwrite(c70, surf_C, (col  )*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c71, surf_C, (col+1)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c72, surf_C, (col+2)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c73, surf_C, (col+3)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c74, surf_C, (col+4)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c75, surf_C, (col+5)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c76, surf_C, (col+6)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c77, surf_C, (col+7)*sizeof(float), row+7  , hipBoundaryModeZero);

}
int main(void)
{
	hipError_t err = hipSuccess;
	
	int m = N;
	int n = N;
	int k = N;
	
	float *A = (float*)malloc(m*k*sizeof(float));
	float *B = (float*)malloc(k*n*sizeof(float));
	float *C = (float*)malloc(m*n*sizeof(float));

	if (A == NULL || B == NULL || C == NULL) {
		printf("allocate host error!\n");
		return 1;
	}

	for (int i = 0; i < m*k; ++i) {
		A[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < k*n; ++i) {
		B[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}


	float *dev_A = NULL;
	float *dev_B = NULL;
	float *dev_C = NULL;

	err = hipMalloc((void**)&dev_A, m*k*sizeof(float));
	err_handling(&err, "allocate devecie error A!");

	err = hipMalloc((void**)&dev_B, k*n*sizeof(float));
	err_handling(&err, "allocate devecie error B!");

	err = hipMalloc((void**)&dev_C, m*n*sizeof(float));
	err_handling(&err, "allocate devecie error C!");
	
	err = hipMemcpy(dev_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to A error!");

	err = hipMemcpy(dev_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to B error!");

	hipChannelFormatDesc ADesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc BDesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc CDesc = hipCreateChannelDesc<float>();
	hipArray *A_array, *B_array, *C_array;
	hipMallocArray(&A_array, &ADesc, k, m);
	hipMallocArray(&B_array, &BDesc, n, k);
	hipMallocArray(&C_array, &CDesc, n, m, hipArraySurfaceLoadStore);
	hipMemcpyToArray(A_array, 0, 0, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(B_array, 0, 0, B, k*n*sizeof(float), hipMemcpyHostToDevice);

	hipBindTextureToArray(tex_A, A_array);
	hipBindTextureToArray(tex_B, B_array);
	cudaBindSurfaceToArray(surf_C, C_array);
	
	tex_A.addressMode[0] = hipAddressModeBorder;
	tex_A.addressMode[1] = hipAddressModeBorder;

	tex_B.addressMode[0] = hipAddressModeBorder;
	tex_B.addressMode[1] = hipAddressModeBorder;



	dim3 dimGrid((n-1)/TILE_WIDTH+1, (m-1)/TILE_WIDTH+1, 1);
	dim3 dimBlock(TILE_WIDTH/WIDTH_PER_THREAD, TILE_WIDTH/WIDTH_PER_THREAD, 1);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	matMul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, m, k, n);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("%fms\n", time_elapsed);

	err = hipMemcpyFromArray(C, C_array, 0, 0, m*n*sizeof(float), hipMemcpyDeviceToHost);
	//err = hipMemcpy(C, dev_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
	err_handling(&err, "memcpy to host C error!");


	FILE *fp = fopen("gpu.out", "w");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fprintf(fp, "%f\n", C[i*N+j]);
		}
	}
	fclose(fp);

	err = hipFree(dev_A);
	err_handling(&err, "mem free A error!");

	err = hipFree(dev_B);
	err_handling(&err, "mem free B error!");

	err = hipFree(dev_C);
	err_handling(&err, "mem free C error!");

	err = hipDeviceReset();
	err_handling(&err, "device reset error!");

	return 0;
}
