#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>

#define TILE_WIDTH 64
#define WIDTH_PER_THREAD 8
#define SW TILE_WIDTH/WIDTH_PER_THREAD
#define N 2048

texture<float, 2, hipReadModeElementType> tex_A;
texture<float, 2, hipReadModeElementType> tex_B;
surface<void, 2> surf_C;

void err_handling(hipError_t *err, const char *str)
{
	if (*err != hipSuccess) {
		printf("%s\n", str);
		exit(EXIT_FAILURE);
	}
}

__global__ void matMul(const float *A, const float *B, float *C, int m, int k, int n)
{
	__shared__ float sA_bf[2][8*64];
	__shared__ float sB_bf[2][8*64];
	float *A_pref, *A_now;
	float *B_pref, *B_now;

	int x = threadIdx.x;
	int y = threadIdx.y;

	int bx = blockIdx.x*64;
	int by = blockIdx.y*64;

	int y8x = y*8+x;
	int x8y = x*8+y;
	int y4x = (y%4)*8+x;

	int idA_global = by + y8x;
	int idB_global = bx + x8y;

	int row = by + y*8;
	int col = bx + x*8;


	float a0, a1, a2, a3, a4, a5, a6, a7;
	float b0, b1, b2, b3, b4, b5, b6, b7;

	float c00 = 0.0; float c01 = 0.0; float c02 = 0.0; float c03 = 0.0, c04 = 0.0; float c05 = 0.0; float c06 = 0.0; float c07 = 0.0;
	float c10 = 0.0; float c11 = 0.0; float c12 = 0.0; float c13 = 0.0, c14 = 0.0; float c15 = 0.0; float c16 = 0.0; float c17 = 0.0;
	float c20 = 0.0; float c21 = 0.0; float c22 = 0.0; float c23 = 0.0, c24 = 0.0; float c25 = 0.0; float c26 = 0.0; float c27 = 0.0;
	float c30 = 0.0; float c31 = 0.0; float c32 = 0.0; float c33 = 0.0, c34 = 0.0; float c35 = 0.0; float c36 = 0.0; float c37 = 0.0;
	float c40 = 0.0; float c41 = 0.0; float c42 = 0.0; float c43 = 0.0, c44 = 0.0; float c45 = 0.0; float c46 = 0.0; float c47 = 0.0;
	float c50 = 0.0; float c51 = 0.0; float c52 = 0.0; float c53 = 0.0, c54 = 0.0; float c55 = 0.0; float c56 = 0.0; float c57 = 0.0;
	float c60 = 0.0; float c61 = 0.0; float c62 = 0.0; float c63 = 0.0, c64 = 0.0; float c65 = 0.0; float c66 = 0.0; float c67 = 0.0;
	float c70 = 0.0; float c71 = 0.0; float c72 = 0.0; float c73 = 0.0, c74 = 0.0; float c75 = 0.0; float c76 = 0.0; float c77 = 0.0;

	sA_bf[0][0*64+y8x] = tex2D(tex_A, 0, idA_global);
	sA_bf[0][1*64+y8x] = tex2D(tex_A, 1, idA_global);
	sA_bf[0][2*64+y8x] = tex2D(tex_A, 2, idA_global);
	sA_bf[0][3*64+y8x] = tex2D(tex_A, 3, idA_global);
	sA_bf[0][4*64+y8x] = tex2D(tex_A, 4, idA_global);
	sA_bf[0][5*64+y8x] = tex2D(tex_A, 5, idA_global);
	sA_bf[0][6*64+y8x] = tex2D(tex_A, 6, idA_global);
	sA_bf[0][7*64+y8x] = tex2D(tex_A, 7, idA_global);

	sB_bf[0][0*64+y8x] = tex2D(tex_B, idB_global, 0);
	sB_bf[0][1*64+y8x] = tex2D(tex_B, idB_global, 1);
	sB_bf[0][2*64+y8x] = tex2D(tex_B, idB_global, 2);
	sB_bf[0][3*64+y8x] = tex2D(tex_B, idB_global, 3);
	sB_bf[0][4*64+y8x] = tex2D(tex_B, idB_global, 4);
	sB_bf[0][5*64+y8x] = tex2D(tex_B, idB_global, 5);
	sB_bf[0][6*64+y8x] = tex2D(tex_B, idB_global, 6);
	sB_bf[0][7*64+y8x] = tex2D(tex_B, idB_global, 7);

	A_pref = sA_bf[1];
	B_pref = sB_bf[1];
	A_now  = sA_bf[0];
	B_now  = sB_bf[0];

	int track_bf = 0;

	for (int t = 8; t < k; t += 8) {

		__syncthreads();

		A_pref[0*64+y8x] = tex2D(tex_A, t  , idA_global);
		A_pref[1*64+y8x] = tex2D(tex_A, t+1, idA_global);
		A_pref[2*64+y8x] = tex2D(tex_A, t+2, idA_global);
		A_pref[3*64+y8x] = tex2D(tex_A, t+3, idA_global);
		A_pref[4*64+y8x] = tex2D(tex_A, t+4, idA_global);
		A_pref[5*64+y8x] = tex2D(tex_A, t+5, idA_global);
		A_pref[6*64+y8x] = tex2D(tex_A, t+6, idA_global);
		A_pref[7*64+y8x] = tex2D(tex_A, t+7, idA_global);

		B_pref[0*64+y8x] = tex2D(tex_B, idB_global, t  );
		B_pref[1*64+y8x] = tex2D(tex_B, idB_global, t+1);
		B_pref[2*64+y8x] = tex2D(tex_B, idB_global, t+2);
		B_pref[3*64+y8x] = tex2D(tex_B, idB_global, t+3);
		B_pref[4*64+y8x] = tex2D(tex_B, idB_global, t+4);
		B_pref[5*64+y8x] = tex2D(tex_B, idB_global, t+5);
		B_pref[6*64+y8x] = tex2D(tex_B, idB_global, t+6);
		B_pref[7*64+y8x] = tex2D(tex_B, idB_global, t+7);



		#pragma unroll
		for (int i = 0; i < 8; ++i) {
			float a = A_now[i*64+y8x];
			float bl = B_now[i*64+y4x];
			float br = B_now[i*64+y4x+32];

			a0 = __shfl(a, 0, 8);
			a1 = __shfl(a, 1, 8);
			a2 = __shfl(a, 2, 8);
			a3 = __shfl(a, 3, 8);
			a4 = __shfl(a, 4, 8);
			a5 = __shfl(a, 5, 8);
			a6 = __shfl(a, 6, 8);
			a7 = __shfl(a, 7, 8);

			b0 = __shfl(bl, x   );
			b1 = __shfl(bl, x+8 );
			b2 = __shfl(bl, x+16);
			b3 = __shfl(bl, x+24);
			b4 = __shfl(br, x   );
			b5 = __shfl(br, x+8 );
			b6 = __shfl(br, x+16);
			b7 = __shfl(br, x+24);
			
			c00 += a0 * b0;
			c01 += a0 * b1;
			c02 += a0 * b2;
			c03 += a0 * b3;
			c04 += a0 * b4;
			c05 += a0 * b5;
			c06 += a0 * b6;
			c07 += a0 * b7;

			c10 += a1 * b0;
			c11 += a1 * b1;
			c12 += a1 * b2;
			c13 += a1 * b3;
			c14 += a1 * b4;
			c15 += a1 * b5;
			c16 += a1 * b6;
			c17 += a1 * b7;

			c20 += a2 * b0;
			c21 += a2 * b1;
			c22 += a2 * b2;
			c23 += a2 * b3;
			c24 += a2 * b4;
			c25 += a2 * b5;
			c26 += a2 * b6;
			c27 += a2 * b7;

			c30 += a3 * b0;
			c31 += a3 * b1;
			c32 += a3 * b2;
			c33 += a3 * b3;
			c34 += a3 * b4;
			c35 += a3 * b5;
			c36 += a3 * b6;
			c37 += a3 * b7;
			
			c40 += a4 * b0;
			c41 += a4 * b1;
			c42 += a4 * b2;
			c43 += a4 * b3;
			c44 += a4 * b4;
			c45 += a4 * b5;
			c46 += a4 * b6;
			c47 += a4 * b7;

			c50 += a5 * b0;
			c51 += a5 * b1;
			c52 += a5 * b2;
			c53 += a5 * b3;
			c54 += a5 * b4;
			c55 += a5 * b5;
			c56 += a5 * b6;
			c57 += a5 * b7;

			c60 += a6 * b0;
			c61 += a6 * b1;
			c62 += a6 * b2;
			c63 += a6 * b3;
			c64 += a6 * b4;
			c65 += a6 * b5;
			c66 += a6 * b6;
			c67 += a6 * b7;

			c70 += a7 * b0;
			c71 += a7 * b1;
			c72 += a7 * b2;
			c73 += a7 * b3;
			c74 += a7 * b4;
			c75 += a7 * b5;
			c76 += a7 * b6;
			c77 += a7 * b7;
		}

		A_pref = sA_bf[track_bf];
		B_pref = sB_bf[track_bf];
		A_now  = sA_bf[1-track_bf];
		B_now  = sB_bf[1-track_bf];
		track_bf = 1 - track_bf;

	}
	__syncthreads();
	#pragma unroll
	for (int i = 0; i < 8; ++i) {
		float a = A_now[i*64+y8x];
		float bl = B_now[i*64+y4x];
		float br = B_now[i*64+y4x+32];

		a0 = __shfl(a, 0, 8);
		a1 = __shfl(a, 1, 8);
		a2 = __shfl(a, 2, 8);
		a3 = __shfl(a, 3, 8);
		a4 = __shfl(a, 4, 8);
		a5 = __shfl(a, 5, 8);
		a6 = __shfl(a, 6, 8);
		a7 = __shfl(a, 7, 8);

		b0 = __shfl(bl, x   );
		b1 = __shfl(bl, x+8 );
		b2 = __shfl(bl, x+16);
		b3 = __shfl(bl, x+24);
		b4 = __shfl(br, x   );
		b5 = __shfl(br, x+8 );
		b6 = __shfl(br, x+16);
		b7 = __shfl(br, x+24);
		
		c00 += a0 * b0;
		c01 += a0 * b1;
		c02 += a0 * b2;
		c03 += a0 * b3;
		c04 += a0 * b4;
		c05 += a0 * b5;
		c06 += a0 * b6;
		c07 += a0 * b7;

		c10 += a1 * b0;
		c11 += a1 * b1;
		c12 += a1 * b2;
		c13 += a1 * b3;
		c14 += a1 * b4;
		c15 += a1 * b5;
		c16 += a1 * b6;
		c17 += a1 * b7;

		c20 += a2 * b0;
		c21 += a2 * b1;
		c22 += a2 * b2;
		c23 += a2 * b3;
		c24 += a2 * b4;
		c25 += a2 * b5;
		c26 += a2 * b6;
		c27 += a2 * b7;

		c30 += a3 * b0;
		c31 += a3 * b1;
		c32 += a3 * b2;
		c33 += a3 * b3;
		c34 += a3 * b4;
		c35 += a3 * b5;
		c36 += a3 * b6;
		c37 += a3 * b7;
		
		c40 += a4 * b0;
		c41 += a4 * b1;
		c42 += a4 * b2;
		c43 += a4 * b3;
		c44 += a4 * b4;
		c45 += a4 * b5;
		c46 += a4 * b6;
		c47 += a4 * b7;

		c50 += a5 * b0;
		c51 += a5 * b1;
		c52 += a5 * b2;
		c53 += a5 * b3;
		c54 += a5 * b4;
		c55 += a5 * b5;
		c56 += a5 * b6;
		c57 += a5 * b7;

		c60 += a6 * b0;
		c61 += a6 * b1;
		c62 += a6 * b2;
		c63 += a6 * b3;
		c64 += a6 * b4;
		c65 += a6 * b5;
		c66 += a6 * b6;
		c67 += a6 * b7;

		c70 += a7 * b0;
		c71 += a7 * b1;
		c72 += a7 * b2;
		c73 += a7 * b3;
		c74 += a7 * b4;
		c75 += a7 * b5;
		c76 += a7 * b6;
		c77 += a7 * b7;
	}



	surf2Dwrite(c00, surf_C, (col  )*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c01, surf_C, (col+1)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c02, surf_C, (col+2)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c03, surf_C, (col+3)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c04, surf_C, (col+4)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c05, surf_C, (col+5)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c06, surf_C, (col+6)*sizeof(float), row  , hipBoundaryModeZero);
	surf2Dwrite(c07, surf_C, (col+7)*sizeof(float), row  , hipBoundaryModeZero);

	surf2Dwrite(c10, surf_C, (col  )*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c11, surf_C, (col+1)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c12, surf_C, (col+2)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c13, surf_C, (col+3)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c14, surf_C, (col+4)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c15, surf_C, (col+5)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c16, surf_C, (col+6)*sizeof(float), row+1  , hipBoundaryModeZero);
	surf2Dwrite(c17, surf_C, (col+7)*sizeof(float), row+1  , hipBoundaryModeZero);

	surf2Dwrite(c20, surf_C, (col  )*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c21, surf_C, (col+1)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c22, surf_C, (col+2)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c23, surf_C, (col+3)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c24, surf_C, (col+4)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c25, surf_C, (col+5)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c26, surf_C, (col+6)*sizeof(float), row+2  , hipBoundaryModeZero);
	surf2Dwrite(c27, surf_C, (col+7)*sizeof(float), row+2  , hipBoundaryModeZero);

	surf2Dwrite(c30, surf_C, (col  )*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c31, surf_C, (col+1)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c32, surf_C, (col+2)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c33, surf_C, (col+3)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c34, surf_C, (col+4)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c35, surf_C, (col+5)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c36, surf_C, (col+6)*sizeof(float), row+3  , hipBoundaryModeZero);
	surf2Dwrite(c37, surf_C, (col+7)*sizeof(float), row+3  , hipBoundaryModeZero);

	surf2Dwrite(c40, surf_C, (col  )*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c41, surf_C, (col+1)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c42, surf_C, (col+2)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c43, surf_C, (col+3)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c44, surf_C, (col+4)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c45, surf_C, (col+5)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c46, surf_C, (col+6)*sizeof(float), row+4  , hipBoundaryModeZero);
	surf2Dwrite(c47, surf_C, (col+7)*sizeof(float), row+4  , hipBoundaryModeZero);

	surf2Dwrite(c50, surf_C, (col  )*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c51, surf_C, (col+1)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c52, surf_C, (col+2)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c53, surf_C, (col+3)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c54, surf_C, (col+4)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c55, surf_C, (col+5)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c56, surf_C, (col+6)*sizeof(float), row+5  , hipBoundaryModeZero);
	surf2Dwrite(c57, surf_C, (col+7)*sizeof(float), row+5  , hipBoundaryModeZero);

	surf2Dwrite(c60, surf_C, (col  )*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c61, surf_C, (col+1)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c62, surf_C, (col+2)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c63, surf_C, (col+3)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c64, surf_C, (col+4)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c65, surf_C, (col+5)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c66, surf_C, (col+6)*sizeof(float), row+6  , hipBoundaryModeZero);
	surf2Dwrite(c67, surf_C, (col+7)*sizeof(float), row+6  , hipBoundaryModeZero);

	surf2Dwrite(c70, surf_C, (col  )*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c71, surf_C, (col+1)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c72, surf_C, (col+2)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c73, surf_C, (col+3)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c74, surf_C, (col+4)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c75, surf_C, (col+5)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c76, surf_C, (col+6)*sizeof(float), row+7  , hipBoundaryModeZero);
	surf2Dwrite(c77, surf_C, (col+7)*sizeof(float), row+7  , hipBoundaryModeZero);

}
int main(void)
{
	hipError_t err = hipSuccess;
	
	int m = N;
	int n = N;
	int k = N;
	
	float *A = (float*)malloc(m*k*sizeof(float));
	float *B = (float*)malloc(k*n*sizeof(float));
	float *C = (float*)malloc(m*n*sizeof(float));

	if (A == NULL || B == NULL || C == NULL) {
		printf("allocate host error!\n");
		return 1;
	}

	for (int i = 0; i < m*k; ++i) {
		A[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < k*n; ++i) {
		B[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}


	float *dev_A = NULL;
	float *dev_B = NULL;
	float *dev_C = NULL;

	err = hipMalloc((void**)&dev_A, m*k*sizeof(float));
	err_handling(&err, "allocate devecie error A!");

	err = hipMalloc((void**)&dev_B, k*n*sizeof(float));
	err_handling(&err, "allocate devecie error B!");

	err = hipMalloc((void**)&dev_C, m*n*sizeof(float));
	err_handling(&err, "allocate devecie error C!");
	
	err = hipMemcpy(dev_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to A error!");

	err = hipMemcpy(dev_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to B error!");

	hipChannelFormatDesc ADesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc BDesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc CDesc = hipCreateChannelDesc<float>();
	hipArray *A_array, *B_array, *C_array;
	hipMallocArray(&A_array, &ADesc, k, m);
	hipMallocArray(&B_array, &BDesc, n, k);
	hipMallocArray(&C_array, &CDesc, n, m, hipArraySurfaceLoadStore);
	hipMemcpyToArray(A_array, 0, 0, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(B_array, 0, 0, B, k*n*sizeof(float), hipMemcpyHostToDevice);

	hipBindTextureToArray(tex_A, A_array);
	hipBindTextureToArray(tex_B, B_array);
	cudaBindSurfaceToArray(surf_C, C_array);
	
	tex_A.addressMode[0] = hipAddressModeBorder;
	tex_A.addressMode[1] = hipAddressModeBorder;

	tex_B.addressMode[0] = hipAddressModeBorder;
	tex_B.addressMode[1] = hipAddressModeBorder;



	dim3 dimGrid((n-1)/TILE_WIDTH+1, (m-1)/TILE_WIDTH+1, 1);
	dim3 dimBlock(TILE_WIDTH/WIDTH_PER_THREAD, TILE_WIDTH/WIDTH_PER_THREAD, 1);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	matMul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, m, k, n);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("%fms\n", time_elapsed);

	err = hipMemcpyFromArray(C, C_array, 0, 0, m*n*sizeof(float), hipMemcpyDeviceToHost);
	//err = hipMemcpy(C, dev_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
	err_handling(&err, "memcpy to host C error!");


	FILE *fp = fopen("gpu.out", "w");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fprintf(fp, "%f\n", C[i*N+j]);
		}
	}
	fclose(fp);

	err = hipFree(dev_A);
	err_handling(&err, "mem free A error!");

	err = hipFree(dev_B);
	err_handling(&err, "mem free B error!");

	err = hipFree(dev_C);
	err_handling(&err, "mem free C error!");

	err = hipDeviceReset();
	err_handling(&err, "device reset error!");

	return 0;
}
