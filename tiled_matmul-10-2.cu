#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>


texture<float, 2, hipReadModeElementType> tex_A;
texture<float, 2, hipReadModeElementType> tex_B;
surface<void, 2> surf_C;

void err_handling(hipError_t *err, const char *str)
{
	if (*err != hipSuccess) {
		printf("%s\n", str);
		exit(EXIT_FAILURE);
	}
}

__global__ void matMul(const float *A, const float *B, float *C, int m, int k, int n)
{
	__shared__ float sA_bf[2][8*64];
	__shared__ float sB_bf[2][8*64];
	float *A_pref, *A_now;
	float *B_pref, *B_now;

	int x = threadIdx.x;
	int y = threadIdx.y;

	int bx = blockIdx.x*64;
	int by = blockIdx.y*64;
	
	int id = y*8+x;
	int inv_id = (id%32)/4*8 + id%4 + (id < 32 ? 0 : 4);
	int glbA_id = by + inv_id;
	int glbB_id = bx + inv_id;

	int row = by + y*8;
	int col = bx + x*8;


	float a[8];
	float b[8];

	float c[8][8];

	for (int i = 0; i < 8; ++i)
		for (int j = 0; j < 8; j++)
			c[i][j] = 0.0;

/************************************************************************/
	
	for (int i = 0; i < 8; ++i) {
		sA_bf[0][i*64+id] = tex2D(tex_A, i, glbA_id);
		sB_bf[0][i*64+id] = tex2D(tex_B, glbB_id, i);
	}

	A_pref = sA_bf[1];
	B_pref = sB_bf[1];
	A_now  = sA_bf[0];
	B_now  = sB_bf[0];

	int track_bf = 0;

	for (int t = 8; t < k; t += 8) {

		__syncthreads();

		#pragma unroll
		for (int i = 0; i < 8; ++i) {
			A_pref[i*64+id] = tex2D(tex_A, t+i, glbA_id);
			B_pref[i*64+id] = tex2D(tex_B, glbB_id, t+i);
			int base = i * 16;

			((float4*)a)[0] = ((float4*)A_now)[base+y];
			((float4*)b)[0] = ((float4*)B_now)[base+x];
			((float4*)a)[1] = ((float4*)A_now)[base+y+8];
			((float4*)b)[1] = ((float4*)B_now)[base+x+8];

			for (int ii = 0; ii < 8; ++ii)
				for (int jj = 0; jj < 8; ++jj)
					c[ii][jj] += a[ii] * b[jj];
		}

		A_pref = sA_bf[track_bf];
		B_pref = sB_bf[track_bf];
		A_now  = sA_bf[1-track_bf];
		B_now  = sB_bf[1-track_bf];
		track_bf = 1 - track_bf;

	}
	__syncthreads();


	#pragma unroll
	for (int i = 0; i < 8; ++i) {
		int base = i * 16;

		((float4*)a)[0] = ((float4*)A_now)[base+y];
		((float4*)b)[0] = ((float4*)B_now)[base+x];
		((float4*)a)[1] = ((float4*)A_now)[base+y+8];
		((float4*)b)[1] = ((float4*)B_now)[base+x+8];

		for (int ii = 0; ii < 8; ++ii)
			for (int jj = 0; jj < 8; ++jj)
				c[ii][jj] += a[ii] * b[jj];
		
	}
	
/******************************** write back ****************************************/
	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < 8; ++j) {
			surf2Dwrite(c[i][j], surf_C, (col+j)*sizeof(float), row+i  , hipBoundaryModeZero);
		}
	}

}
int main(int argc, char *argv[])
{
	if (argc != 4) {
		printf("usage: ./xxx m n k\n");
		return -1;
	}
	
	hipError_t err = hipSuccess;
	
	int m = atoi(argv[1]);
	int n = atoi(argv[2]);
	int k = atoi(argv[3]);
	
	float *A = (float*)malloc(m*k*sizeof(float));
	float *B = (float*)malloc(k*n*sizeof(float));
	float *C = (float*)malloc(m*n*sizeof(float));

	if (A == NULL || B == NULL || C == NULL) {
		printf("allocate host error!\n");
		return 1;
	}

	for (int i = 0; i < m*k; ++i) {
		A[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < k*n; ++i) {
		B[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}


	float *dev_A = NULL;
	float *dev_B = NULL;
	float *dev_C = NULL;

	err = hipMalloc((void**)&dev_A, m*k*sizeof(float));
	err_handling(&err, "allocate devecie error A!");

	err = hipMalloc((void**)&dev_B, k*n*sizeof(float));
	err_handling(&err, "allocate devecie error B!");

	err = hipMalloc((void**)&dev_C, m*n*sizeof(float));
	err_handling(&err, "allocate devecie error C!");
	
	err = hipMemcpy(dev_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to A error!");

	err = hipMemcpy(dev_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to B error!");

	hipChannelFormatDesc ADesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc BDesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc CDesc = hipCreateChannelDesc<float>();
	hipArray *A_array, *B_array, *C_array;
	hipMallocArray(&A_array, &ADesc, k, m);
	hipMallocArray(&B_array, &BDesc, n, k);
	hipMallocArray(&C_array, &CDesc, n, m, hipArraySurfaceLoadStore);
	hipMemcpyToArray(A_array, 0, 0, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(B_array, 0, 0, B, k*n*sizeof(float), hipMemcpyHostToDevice);

	hipBindTextureToArray(tex_A, A_array);
	hipBindTextureToArray(tex_B, B_array);
	cudaBindSurfaceToArray(surf_C, C_array);
	
	tex_A.addressMode[0] = hipAddressModeBorder;
	tex_A.addressMode[1] = hipAddressModeBorder;

	tex_B.addressMode[0] = hipAddressModeBorder;
	tex_B.addressMode[1] = hipAddressModeBorder;



	dim3 dimGrid((n-1)/64+1, (m-1)/64+1, 1);
	dim3 dimBlock(8, 8, 1);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	matMul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, m, k, n);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("%fms\n", time_elapsed);

	err = hipMemcpyFromArray(C, C_array, 0, 0, m*n*sizeof(float), hipMemcpyDeviceToHost);
	//err = hipMemcpy(C, dev_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
	err_handling(&err, "memcpy to host C error!");


	FILE *fp = fopen("gpu.out", "w");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fprintf(fp, "%f\n", C[i*n+j]);
		}
	}
	fclose(fp);

	err = hipFree(dev_A);
	err_handling(&err, "mem free A error!");

	err = hipFree(dev_B);
	err_handling(&err, "mem free B error!");

	err = hipFree(dev_C);
	err_handling(&err, "mem free C error!");

	err = hipDeviceReset();
	err_handling(&err, "device reset error!");

	return 0;
}
