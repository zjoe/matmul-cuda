#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>

#define TILE_WIDTH 16
#define N 2048


void err_handling(hipError_t *err, const char *str)
{
	if (*err != hipSuccess) {
		printf("%s\n", str);
		exit(EXIT_FAILURE);
	}
}

__global__ void matMul(const float *A, const float *B, float *C, int m, int k, int n)
{
	__shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

	//int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int row = blockIdx.y*blockDim.y + ty;
	int col = blockIdx.x*blockDim.x + tx;

	float cVal = 0.0;
	
	for (int t = 0; t < k/TILE_WIDTH; ++t) {
		sh_A[ty][tx] = A[row*k + t*TILE_WIDTH + tx];
		sh_B[ty][tx] = B[(t*TILE_WIDTH + ty)*k + col];
		__syncthreads();

		for (int i = 0; i < TILE_WIDTH; ++i) {
			cVal += sh_A[ty][i] * sh_B[i][tx];
		}
		__syncthreads();
	}
	C[row*n + col] = cVal;
}

int main(void)
{
	hipError_t err = hipSuccess;
	
	int m = N;
	int n = N;
	int k = N;
	
	float *A = (float*)malloc(m*k*sizeof(float));
	float *B = (float*)malloc(k*n*sizeof(float));
	float *C = (float*)malloc(m*n*sizeof(float));

	if (A == NULL || B == NULL || C == NULL) {
		printf("allocate host error!\n");
		return 1;
	}

	for (int i = 0; i < m*k; ++i) {
		A[i] = rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < k*n; ++i) {
		B[i] = rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < m*n; ++i) {
		C[i] = rand()/(float)RAND_MAX;
	}

	float *dev_A = NULL;
	float *dev_B = NULL;
	float *dev_C = NULL;

	err = hipMalloc((void**)&dev_A, m*k*sizeof(float));
	err_handling(&err, "allocate devecie error A!");

	err = hipMalloc((void**)&dev_B, k*n*sizeof(float));
	err_handling(&err, "allocate devecie error B!");

	err = hipMalloc((void**)&dev_C, m*n*sizeof(float));
	err_handling(&err, "allocate devecie error C!");
	
	err = hipMemcpy(dev_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to A error!");

	err = hipMemcpy(dev_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to B error!");

	dim3 dimGrid((m-1)/TILE_WIDTH+1, (n-1)/TILE_WIDTH+1, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	matMul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, m, k, n);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("%fms\n", time_elapsed);

	err = hipMemcpy(C, dev_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
	err_handling(&err, "memcpy to host C error!");

	printf("%f %f\n", C[100*N+100], C[234*N+234]);

	err = hipFree(dev_A);
	err_handling(&err, "mem free A error!");

	err = hipFree(dev_B);
	err_handling(&err, "mem free B error!");

	err = hipFree(dev_C);
	err_handling(&err, "mem free C error!");

	err = hipDeviceReset();
	err_handling(&err, "device reset error!");

	return 0;
}
