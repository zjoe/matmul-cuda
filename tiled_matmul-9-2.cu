#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>


texture<float, 2, hipReadModeElementType> tex_A;
texture<float, 2, hipReadModeElementType> tex_B;
surface<void, 2> surf_C;

void err_handling(hipError_t *err, const char *str)
{
	if (*err != hipSuccess) {
		printf("%s\n", str);
		exit(EXIT_FAILURE);
	}
}

__global__ void matMul(const float *A, const float *B, float *C, int m, int k, int n)
{
	__shared__ float sA_bf[2][8*64];
	__shared__ float sB_bf[2][8*64];
	float *A_pref, *A_now;
	float *B_pref, *B_now;

	int x = threadIdx.x;
	int y = threadIdx.y;

	int bx = blockIdx.x*64;
	int by = blockIdx.y*64;
	
	int id = y*8+x;
	int inv_id = (id%32)/4*8 + id%4 + (id < 32 ? 0 : 4);
	int glbA_id = by + inv_id;
	int glbB_id = bx + inv_id;

	int row = by + y*8;
	int col = bx + x*8;


	float a0[8];
	float a1[8];
	float b0[8];
	float b1[8];

	float c[8][8];

	for (int i = 0; i < 8; ++i)
		for (int j = 0; j < 8; j++)
			c[i][j] = 0.0;
	
/*********************************************************************/
	for (int i = 0; i < 8; ++i) { // first batch of shared store
		sA_bf[0][i*64+id] = tex2D(tex_A, i, glbA_id);
		sB_bf[0][i*64+id] = tex2D(tex_B, glbB_id, i);
	}

	A_pref = sA_bf[1];
	B_pref = sB_bf[1];
	A_now  = sA_bf[0];
	B_now  = sB_bf[0];

	int track_bf = 0;

/****************************** main loop ******************************/
	for (int t = 8; t < k; t += 8) {

		__syncthreads();

		A_pref[id] = tex2D(tex_A, t, glbA_id); // double buffered shared store
		B_pref[id] = tex2D(tex_B, glbB_id, t);

		((float4*)a0)[0] = ((float4*)A_now)[y]; // first shared load of each step
		((float4*)b0)[0] = ((float4*)B_now)[x];
		((float4*)a0)[1] = ((float4*)A_now)[y+8];
		((float4*)b0)[1] = ((float4*)B_now)[x+8];
		
		#pragma unroll
		for (int i = 1; i < 8; ++i) {
			int base = i * 16;
			A_pref[i*64+id] = tex2D(tex_A, t+i, glbA_id); // double bufferd shared store
			B_pref[i*64+id] = tex2D(tex_B, glbB_id, t+i);

			if (i&1) {
				((float4*)a1)[0] = ((float4*)A_now)[base+y]; // double buffered shared load
				((float4*)b1)[0] = ((float4*)B_now)[base+x];
				((float4*)a1)[1] = ((float4*)A_now)[base+y+8];
				((float4*)b1)[1] = ((float4*)B_now)[base+x+8];

				for (int ii = 0; ii < 8; ++ii)
					for (int jj = 0; jj < 8; ++jj)
						c[ii][jj] += a0[ii] * b0[jj];
				
			} else {
				((float4*)a0)[0] = ((float4*)A_now)[base+y]; // double buffered shared load
				((float4*)b0)[0] = ((float4*)B_now)[base+x];
				((float4*)a0)[1] = ((float4*)A_now)[base+y+8];
				((float4*)b0)[1] = ((float4*)B_now)[base+x+8];

				for (int ii = 0; ii < 8; ++ii)
					for (int jj = 0; jj < 8; ++jj)
						c[ii][jj] += a1[ii] * b1[jj];

			}
		}

		for (int i = 0; i < 8; ++i) { // remained computation of each step
			for (int j = 0; j < 8; ++j) {
				c[i][j] += a1[i] * b1[j];
			}
		}

		A_pref = sA_bf[track_bf]; // shared double buffer pointer exchange
		B_pref = sB_bf[track_bf];
		A_now  = sA_bf[1-track_bf];
		B_now  = sB_bf[1-track_bf];
		track_bf = 1 ^ track_bf; // flip between 0 & 1

	}
	__syncthreads(); // need sync to ensure the last shared store complete

/************************************ remained step *******************************************/

	((float4*)a0)[0] = ((float4*)A_now)[y];
	((float4*)b0)[0] = ((float4*)B_now)[x];
	((float4*)a0)[1] = ((float4*)A_now)[y+8];
	((float4*)b0)[1] = ((float4*)B_now)[x+8];

	#pragma unroll
	for (int i = 1; i < 8; ++i) {
		int base = i * 16;

		if (i&1) {
			((float4*)a1)[0] = ((float4*)A_now)[base+y];
			((float4*)b1)[0] = ((float4*)B_now)[base+x];
			((float4*)a1)[1] = ((float4*)A_now)[base+y+8];
			((float4*)b1)[1] = ((float4*)B_now)[base+x+8];

			for (int ii = 0; ii < 8; ++ii)
				for (int jj = 0; jj < 8; ++jj)
					c[ii][jj] += a0[ii] * b0[jj];

		} else {
			((float4*)a0)[0] = ((float4*)A_now)[base+y];
			((float4*)b0)[0] = ((float4*)B_now)[base+x];
			((float4*)a0)[1] = ((float4*)A_now)[base+y+8];
			((float4*)b0)[1] = ((float4*)B_now)[base+x+8];

			for (int ii = 0; ii < 8; ++ii)
				for (int jj = 0; jj < 8; ++jj)
					c[ii][jj] += a1[ii] * b1[jj];

		}

	}

	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < 8; ++j) {
			c[i][j] += a1[i] * b1[j];
		}
	}

/********************************** wirte back *****************************************/
	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < 8; ++j) {
			surf2Dwrite(c[i][j], surf_C, (col+j)*sizeof(float), row+i, hipBoundaryModeZero);
		}
	}

}
int main(int argc, char *argv[])
{
	if (argc != 4) {
		printf("usage: ./xxx m n k\n");
		return -1;
	}
	
	hipError_t err = hipSuccess;
	
	int m = atoi(argv[1]);
	int n = atoi(argv[2]);
	int k = atoi(argv[3]);
	
	float *A = (float*)malloc(m*k*sizeof(float));
	float *B = (float*)malloc(k*n*sizeof(float));
	float *C = (float*)malloc(m*n*sizeof(float));

	if (A == NULL || B == NULL || C == NULL) {
		printf("allocate host error!\n");
		return 1;
	}

	for (int i = 0; i < m*k; ++i) {
		A[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}

	for (int i = 0; i < k*n; ++i) {
		B[i] = rand()/(float)RAND_MAX - rand()/(float)RAND_MAX;
	}


	float *dev_A = NULL;
	float *dev_B = NULL;
	float *dev_C = NULL;

	err = hipMalloc((void**)&dev_A, m*k*sizeof(float));
	err_handling(&err, "allocate devecie error A!");

	err = hipMalloc((void**)&dev_B, k*n*sizeof(float));
	err_handling(&err, "allocate devecie error B!");

	err = hipMalloc((void**)&dev_C, m*n*sizeof(float));
	err_handling(&err, "allocate devecie error C!");
	
	err = hipMemcpy(dev_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to A error!");

	err = hipMemcpy(dev_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
	err_handling(&err, "memcpy to B error!");

	hipChannelFormatDesc ADesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc BDesc = hipCreateChannelDesc<float>();
	hipChannelFormatDesc CDesc = hipCreateChannelDesc<float>();
	hipArray *A_array, *B_array, *C_array;
	hipMallocArray(&A_array, &ADesc, k, m);
	hipMallocArray(&B_array, &BDesc, n, k);
	hipMallocArray(&C_array, &CDesc, n, m, hipArraySurfaceLoadStore);
	hipMemcpyToArray(A_array, 0, 0, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToArray(B_array, 0, 0, B, k*n*sizeof(float), hipMemcpyHostToDevice);

	hipBindTextureToArray(tex_A, A_array);
	hipBindTextureToArray(tex_B, B_array);
	cudaBindSurfaceToArray(surf_C, C_array);
	
	tex_A.addressMode[0] = hipAddressModeBorder;
	tex_A.addressMode[1] = hipAddressModeBorder;

	tex_B.addressMode[0] = hipAddressModeBorder;
	tex_B.addressMode[1] = hipAddressModeBorder;



	dim3 dimGrid((n-1)/64+1, (m-1)/64+1, 1);
	dim3 dimBlock(8, 8, 1);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	matMul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, m, k, n);
	hipEventRecord(stop, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	float time_elapsed = 0;
	hipEventElapsedTime(&time_elapsed, start, stop);
	printf("%fms\n", time_elapsed);

	err = hipMemcpyFromArray(C, C_array, 0, 0, m*n*sizeof(float), hipMemcpyDeviceToHost);
	//err = hipMemcpy(C, dev_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
	err_handling(&err, "memcpy to host C error!");


	FILE *fp = fopen("gpu.out", "w");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			fprintf(fp, "%f\n", C[i*n+j]);
		}
	}
	fclose(fp);

	err = hipFree(dev_A);
	err_handling(&err, "mem free A error!");

	err = hipFree(dev_B);
	err_handling(&err, "mem free B error!");

	err = hipFree(dev_C);
	err_handling(&err, "mem free C error!");

	err = hipDeviceReset();
	err_handling(&err, "device reset error!");

	return 0;
}
